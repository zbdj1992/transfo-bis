#include "hip/hip_runtime.h"
/**
* nvidia_gpu.cpp
* Author: zhubin
* Created on: 2019-02-12
* Copyright (c) luchangzhineng. All Rights Reserved
*/

#include "nvidia_gpu.h"
#include "hip/hip_runtime.h"

namespace transfo_bis {

NvidiaGpu::NvidiaGpu()
{
	hipFree(0);
	hipMalloc((void**)&_1Mfloat_aa, 1024 * 1024 * sizeof(float));
	hipMalloc((void**)&_1Mfloat_bb, 1024 * 1024 * sizeof(float));
	hipMalloc((void**)&_1Mfloat_cc, 1024 * 1024 * sizeof(float));
	hipMalloc((void**)&_ibeoSrcImg, 512*512*sizeof(unsigned char));
	hipMalloc((void**)&_ibeoDstImg, 512*512*sizeof(unsigned char));
	hipMalloc((void**)&_ibeoTmpImg, 512*512*sizeof(unsigned char));
	
}

NvidiaGpu::~NvidiaGpu()
{
	hipFree(_1Mfloat_aa);
	hipFree(_1Mfloat_bb);
	hipFree(_1Mfloat_cc);
	hipFree(_ibeoSrcImg);
	hipFree(_ibeoDstImg);
	hipFree(_ibeoTmpImg);
}

NvidiaGpu* NvidiaGpu::get_obj()
{
	static NvidiaGpu tmp_obj;
	return &tmp_obj;
}

__global__ void KeAddVecVecFloat(float *a, float *b, float *c, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n) {
		c[i] = a[i] + b[i];
	}
}
__global__ void KeImgCopy(unsigned char *src,unsigned char *dst,int width,int height){
	int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
	if((xIndex < width) && (yIndex < height)){
		dst[yIndex*width+xIndex] = src[yIndex*width+xIndex];
	}
}
__global__ void KeImgRev(unsigned char *img,int width,int height){
	int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
	if((xIndex < width) && (yIndex < height)){
		if(img[yIndex*width+xIndex] > 0){
			img[yIndex*width+xIndex] = 0;
		}else{
			img[yIndex*width+xIndex] = 255;
		}
	}
}

__global__ void KeImgRangeConvStep1(unsigned char *img,int width,int height,int flag){
	int m1 = 3;
	int m3 = m1;
	int m2 = 1;
	int m4 = 3;
	if (flag==0) {
		m1 = 1;m3 = 1;
		m2 = 1;m4 = 1;
	}
	if (flag==2) {
		m2 = 3;m4 = 1;
		m1 = 3;m3 = 3;
	}
	int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
	if((xIndex > 0) && (yIndex > 0) && (xIndex < width-1) && (yIndex < height-1)){
		if(img[yIndex*width+xIndex] < 254){
			int xIndexTmp = xIndex-1;
			int yIndexTmp = yIndex+1;
			if((img[yIndex*width+xIndex]+m1) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndex*width+xIndex]+m1;
			xIndexTmp = xIndex;
			yIndexTmp = yIndex+1;
			if((img[yIndex*width+xIndex]+m2) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndex*width+xIndex]+m2;
			xIndexTmp = xIndex+1;
			yIndexTmp = yIndex+1;
			if((img[yIndex*width+xIndex]+m3) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndex*width+xIndex]+m3;
			xIndexTmp = xIndex+1;
			yIndexTmp = yIndex;
			if((img[yIndex*width+xIndex]+m4) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndex*width+xIndex]+m4;

		}
	}
}
__global__ void KeImgRangeConvStep2(unsigned char *img,int width,int height,int flag){
	int m1 = 3;
	int m3 = m1;
	int m2 = 1;
	int m4 = 3;
	if (flag==0) {
		m1 = 1;m3 = 1;
		m2 = 1;m4 = 1;
	}
	if (flag==2) {
		m2 = 3;m4 = 1;
		m1 = 3;m3 = 3;
	}
	int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
	int xIndexR = width - xIndex;
	int yIndexR = height - yIndex;
	if((xIndexR > 0) && (yIndexR > 0) && (xIndexR < width-2) && (yIndexR < height-2)){
		if(img[yIndexR*width+xIndexR] < 254){
			int xIndexTmp = xIndexR-1;
			int yIndexTmp = yIndexR-1;
			if((img[yIndexR*width+xIndexR]+m1) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndexR*width+xIndexR]+m1;
			xIndexTmp = xIndexR;
			yIndexTmp = yIndexR-1;
			if((img[yIndexR*width+xIndexR]+m2) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndexR*width+xIndexR]+m2;
			xIndexTmp = xIndexR+1;
			yIndexTmp = yIndexR-1;
			if((img[yIndexR*width+xIndexR]+m3) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndexR*width+xIndexR]+m3;
			xIndexTmp = xIndexR-1;
			yIndexTmp = yIndexR;
			if((img[yIndexR*width+xIndexR]+m4) < img[yIndexTmp*width+xIndexTmp])
				img[yIndexTmp*width+xIndexTmp] = img[yIndexR*width+xIndexR]+m4;

		}
	}
}
__global__ void KeImgConvMassPoint(unsigned char *img,int width,int height,int scale){
	int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
	if((xIndex < width) && (yIndex < height)){
		if(img[yIndex*width+xIndex] <= scale)
			img[yIndex*width+xIndex] = 255;
	    else
	    	img[yIndex*width+xIndex] = 0;
	    }
}

int NvidiaGpu::add_vec_vec_float(float *a, float *b, float *c,
	int num1, int num2, int num3)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (num1 != num2 || num1 != num3) {
		return -1;
	}
	if ((num1 >> 20) > 0) {
		printf("size to big!\n");
		return -1;
	}
	int block_size = 512;
	int grid_size = num1 >> 9;
	grid_size += 1;
	dim3 dimBlock(block_size);
	dim3 dimGrid(grid_size);
	int byte_size = num1 * sizeof(float);
	hipMemcpy(_1Mfloat_aa, a, byte_size, hipMemcpyHostToDevice);
	hipMemcpy(_1Mfloat_bb, b, byte_size, hipMemcpyHostToDevice);
	hipEventRecord(start);
	KeAddVecVecFloat<<<dimGrid, dimBlock>>>(_1Mfloat_aa, _1Mfloat_bb, _1Mfloat_cc, num1);
	hipEventRecord(stop);
	hipMemcpy(c, _1Mfloat_cc, byte_size, hipMemcpyDeviceToHost);

	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("cuda time:[%.2f]ms\n", time);
	return 0;
}
int NvidiaGpu::gen_drive_area(unsigned char* src,unsigned char* dst,int width,int height)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int byteSize = width*height;
	int scale = 20;
	int flag = 1;
	hipMemcpy(_ibeoSrcImg,src,byteSize,hipMemcpyHostToDevice);
	dim3 threadsPerBlock(32,32);
	dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,(height + threadsPerBlock.y -1)/ threadsPerBlock.y);
	hipEventRecord(start);
	KeImgCopy<<<blocksPerGrid,threadsPerBlock>>> (_ibeoSrcImg,_ibeoTmpImg,width,height);
	KeImgRev<<<blocksPerGrid,threadsPerBlock>>> (_ibeoTmpImg,width,height);
	KeImgRangeConvStep1<<<blocksPerGrid,threadsPerBlock>>> (_ibeoTmpImg,width,height,flag);
	KeImgRangeConvStep2<<<blocksPerGrid,threadsPerBlock>>> (_ibeoTmpImg,width,height,flag);
	KeImgConvMassPoint<<<blocksPerGrid,threadsPerBlock>>> (_ibeoTmpImg,width,height,scale);
	hipEventRecord(stop);
	//
	hipMemcpy(dst,_ibeoTmpImg,byteSize,hipMemcpyDeviceToHost);
	//hipMemcpy(dst,_ibeoDstImg,byteSize,hipMemcpyDeviceToHost);
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("cuda time:[%.2f]ms\n", time);
	return 0;
}
}
